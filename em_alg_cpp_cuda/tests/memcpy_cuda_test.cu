#include <iostream>
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>

int main() {
    unsigned long size = 1000 * 100 * 20 * 3;

    std::vector<unsigned> v(size, 999);

    unsigned* devicePtr;

    auto res = hipMalloc(&devicePtr, size * sizeof(unsigned));
    if (res != hipSuccess) {
        std::cerr << "could not alloc memory" << std::endl;
        return 1;
    }

    auto now1 = std::chrono::system_clock::now();
    hipMemcpy(devicePtr, v.data(), size * sizeof(unsigned), hipMemcpyHostToDevice);
    auto now2 = std::chrono::system_clock::now();

    std::cout << "coping to device took " << std::chrono::duration_cast<std::chrono::milliseconds>(now2 - now1).count() << std::endl;

    now1 = std::chrono::system_clock::now();
    hipMemcpy(v.data(), devicePtr, size * sizeof(unsigned), hipMemcpyDeviceToHost);
    now2 = std::chrono::system_clock::now();

    std::cout << "coping to host took " << std::chrono::duration_cast<std::chrono::milliseconds>(now2 - now1).count() << std::endl;

    return 0;
}