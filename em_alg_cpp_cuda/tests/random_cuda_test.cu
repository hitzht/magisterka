#include <iostream>
#include <array>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include <hip/hip_runtime_api.h>

__global__ void kernel(unsigned* data, hiprandState* randomState, unsigned long seed) {
    unsigned id = threadIdx.x;
    hiprand_init(seed, id, 0, randomState);

    for (unsigned i = 0; i < 10; i++) {
        data[i] = 99 * hiprand_uniform(randomState);
    }
}

int main() {
    unsigned numbers[10];

    unsigned* deviceNumbers;

    auto result = hipMalloc(&deviceNumbers, sizeof(unsigned) * 10);
    if (result != hipSuccess) {
        std::cerr << "could not alloc memory!" << std::endl;
        return 1;
    }

    hiprandState* randomState;
    result = hipMalloc(&randomState, sizeof(hiprandState));
    if (result != hipSuccess) {
        std::cerr << "could not alloc random state!" << std::endl;
        return 1;
    }

    unsigned long seed = time(nullptr);
    kernel<<<1, 1>>>(deviceNumbers, randomState, seed);

    hipMemcpy(numbers, deviceNumbers, sizeof(unsigned) * 10, hipMemcpyDeviceToHost);

    for (auto val : numbers) {
        std::cout << val << std::endl;
    }

    hipFree(deviceNumbers);
    hipFree(randomState);

    return 0;
}
