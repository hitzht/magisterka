#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "../src/gpu/PMX.h"


__global__ void kernel(unsigned dimension, unsigned* p1, unsigned* p2, unsigned start, unsigned end, unsigned* result) {
    pmx(dimension, p1, p2, start, end, result);
}

void test1() {
    unsigned p1[]{8, 4, 7, 3, 6, 2, 5, 1, 9, 0};
    unsigned p2[]{0, 1, 2, 3, 4, 5, 6, 7, 8, 9};

    unsigned * deviceP1;
    unsigned * deviceP2;
    unsigned * deviceResult;

    hipMalloc(&deviceP1, sizeof(p1));
    hipMalloc(&deviceP2, sizeof(p2));
    hipMalloc(&deviceResult, sizeof(p2));

    hipMemcpy(deviceP1, p1, sizeof(p1), hipMemcpyHostToDevice);
    hipMemcpy(deviceP2, p2, sizeof(p2), hipMemcpyHostToDevice);


    kernel<<<1, 1>>>(10, deviceP1, deviceP2, 3, 7, deviceResult);

    unsigned result[10];

    hipMemcpy(p1, deviceP1, sizeof(p2), hipMemcpyDeviceToHost);
    hipMemcpy(p2, deviceP2, sizeof(p2), hipMemcpyDeviceToHost);
    hipMemcpy(result, deviceResult, sizeof(p2), hipMemcpyDeviceToHost);

    for (auto val : p1)
        std::cout << val << " ";

    std::cout << std::endl;

    for (auto val : p2)
        std::cout << val << " ";

    std::cout << std::endl;

    for (auto val : result) // 0, 7, 4, 3, 6, 2, 5, 1, 8, 9
        std::cout << val << " ";

    std::cout << std::endl;

    hipFree(deviceP1);
    hipFree(deviceP2);
    hipFree(deviceResult);
}

void test2() {
    unsigned p1[]{1, 2, 3, 4, 5, 6, 7, 8, 9};
    unsigned p2[]{4, 5, 2, 1, 8, 7, 6, 9, 3};

    unsigned * deviceP1;
    unsigned * deviceP2;
    unsigned * deviceResult;

    hipMalloc(&deviceP1, sizeof(p1));
    hipMalloc(&deviceP2, sizeof(p2));
    hipMalloc(&deviceResult, sizeof(p2));

    hipMemcpy(deviceP1, p1, sizeof(p1), hipMemcpyHostToDevice);
    hipMemcpy(deviceP2, p2, sizeof(p2), hipMemcpyHostToDevice);


    kernel<<<1, 1>>>(9, deviceP1, deviceP2, 3, 6, deviceResult);

    unsigned result[9];

    hipMemcpy(p1, deviceP1, sizeof(p2), hipMemcpyDeviceToHost);
    hipMemcpy(p2, deviceP2, sizeof(p2), hipMemcpyDeviceToHost);
    hipMemcpy(result, deviceResult, sizeof(p2), hipMemcpyDeviceToHost);

    for (auto val : p1)
        std::cout << val << " ";

    std::cout << std::endl;

    for (auto val : p2)
        std::cout << val << " ";

    std::cout << std::endl;

    for (auto val : result) // 1, 8, 2, 4, 5, 6, 7, 9, 3
        std::cout << val << " ";

    std::cout << std::endl;

    hipFree(deviceP1);
    hipFree(deviceP2);
    hipFree(deviceResult);
}

void test3() {
    unsigned p1[]{1, 5, 2, 8, 7, 4, 3, 6};
    unsigned p2[]{4, 2, 5, 8, 1, 3, 6, 7};

    unsigned * deviceP1;
    unsigned * deviceP2;
    unsigned * deviceResult;

    hipMalloc(&deviceP1, sizeof(p1));
    hipMalloc(&deviceP2, sizeof(p2));
    hipMalloc(&deviceResult, sizeof(p2));

    hipMemcpy(deviceP1, p1, sizeof(p1), hipMemcpyHostToDevice);
    hipMemcpy(deviceP2, p2, sizeof(p2), hipMemcpyHostToDevice);


    kernel<<<1, 1>>>(8, deviceP1, deviceP2, 2, 4, deviceResult);

    unsigned result[8];

    hipMemcpy(p1, deviceP1, sizeof(p2), hipMemcpyDeviceToHost);
    hipMemcpy(p2, deviceP2, sizeof(p2), hipMemcpyDeviceToHost);
    hipMemcpy(result, deviceResult, sizeof(p2), hipMemcpyDeviceToHost);

    for (auto val : p1)
        std::cout << val << " ";

    std::cout << std::endl;

    for (auto val : p2)
        std::cout << val << " ";

    std::cout << std::endl;

    for (auto val : result) // 4, 5, 2, 8, 7, 3, 6, 1
        std::cout << val << " ";

    std::cout << std::endl;

    hipFree(deviceP1);
    hipFree(deviceP2);
    hipFree(deviceResult);
}

int main() {
    test1();
    test2();
    test3();

    return 0;
}