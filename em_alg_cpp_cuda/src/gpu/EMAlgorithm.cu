#include "EMAlgorithm.h"
#include <stdexcept>
#include <algorithm>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "GPU.hpp"

unsigned EMAlgorithm::solve(const AlgorithmInput &input) {
    auto permutationsCount = input.permutations.size();
    std::vector<unsigned> calculatedValues(permutationsCount);

    hiprandState* randomStates;
    auto result = hipMalloc(&randomStates, sizeof(hiprandState) * permutationsCount);

    if (result != hipSuccess)
        throw std::runtime_error("Error while allocating random states, error code: " + std::to_string(result));

    unsigned long seed = time(nullptr);

    initializeRandomStates<<<input.blocks, input.threads>>>(randomStates, permutationsCount, seed);

    auto deviceWeights = allocateData(input.weights);
    auto deviceDistances = allocateData(input.distances);
    auto devicePermutations = allocateData(input.permutations);
    auto deviceValues = allocateArray(permutationsCount);
    auto deviceNextPermutations = allocateArray(permutationsCount * input.dimension);
    auto pmxBuffer = allocateArray(permutationsCount * input.dimension);

    unsigned bestPermutationValue{0};

    for (unsigned iteration = 0; iteration < input.iterations; iteration++) {
        std::cout << "iter " << iteration << std::endl;

        calculateQAPValues<<<input.blocks, input.threads>>>(input.dimension, permutationsCount, deviceWeights,
                                                           deviceDistances, devicePermutations, deviceValues);

        performMovement<<<input.blocks, input.threads>>>(input.dimension, permutationsCount, input.neighborhoodDistance,
                                                         devicePermutations, deviceValues, deviceNextPermutations, pmxBuffer, randomStates);

        copyPermutations<<<input.blocks, input.threads>>>(input.dimension, permutationsCount, devicePermutations, deviceNextPermutations);
    }

    result = hipMemcpy(calculatedValues.data(), deviceValues, sizeof(unsigned) * permutationsCount, hipMemcpyDeviceToHost);
    if (result != hipSuccess)
        throw std::runtime_error{"Error while coping output to host, error code: " + std::to_string(result)};

    bestPermutationValue = *std::min_element(calculatedValues.begin(), calculatedValues.end());

    hipFree(deviceWeights);
    hipFree(deviceDistances);
    hipFree(devicePermutations);
    hipFree(deviceValues);
    hipFree(deviceNextPermutations);
    hipFree(randomStates);
    hipFree(pmxBuffer);

    return bestPermutationValue;
}
